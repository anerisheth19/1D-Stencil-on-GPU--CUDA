//ECGR 6090 Heterogeneous Computing Homework 0
// Problem 2 c - 1D Stencil on GPU with shared memory
//Written by Aneri Sheth - 801085402

// Reference taken from Lecture Slides by Dr. Tabkhi 
//Other reference taken from https://github.com/szymonm/pwir-cuda-labs/tree/master/lab1 and https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#using-cuda-gpu-timers



#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

#define RADIUS 2 //radius = 2,4,8,16
#define BLOCK_SIZE    128 //fixed number of threads per block 
#define NUM_ELEMENTS 1000 //job size = 1K, 10K, 100K, 1M and 10M


// CUDA API error checking macro
static void handleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define cudaCheck( err ) (handleError( err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in, int *out) 
{
	__shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
	int gindex = threadIdx.x + blockIdx.x * blockDim.x;
	int lindex = threadIdx.x + RADIUS;
	temp[lindex] = in[gindex]; //storing in shared memory
	
	if (threadIdx.x < RADIUS) 
	{
	temp[lindex - RADIUS] = in[gindex - RADIUS];
	temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
	}
	__syncthreads();
	int result = 0;
	for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
	{
		result += temp[lindex + offset];
	}
// Store the result
	out[gindex] = result;
}	

int main()
{
  unsigned int i;
  //CPU array copies
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];
//GPU array copies
  int *d_in, *d_out;

	hipEvent_t start, stop; //time start and stop
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i )
    h_in[i] = 1; 

  // Allocate device memory
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  //copy fro CPU to GPU memory
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), hipMemcpyHostToDevice) );
  hipEventRecord( start, 0 );

 //Call stencil kernel
  stencil_1d<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);

   	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf("GPU Execution Time = %f\n",time);

  //copy from device to host
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), hipMemcpyDeviceToHost) );

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
