//ECGR 6090 Heterogeneous Computing Homework 0
// Problem 2 a - 1D Stencil on GPU
//Written by Aneri Sheth - 801085402

// Reference taken from Lecture Slides by Dr. Tabkhi 
//Other reference taken from https://github.com/szymonm/pwir-cuda-labs/tree/master/lab1 and https://docs.nvidia.com/cuda/cuda-c-best-practices-guide/index.html#using-cuda-gpu-timers




#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>


#define RADIUS        2 //radius = 2,4,8,16
#define BLOCK_SIZE    128 //fixed number of threads per block 
#define NUM_ELEMENTS  10000 //job size = 1K, 10K, 100K, 1M and 10M

// CUDA API error checking macro
static void handleError(hipError_t err,
                        const char *file,
                        int line ) {
    if (err != hipSuccess) {
        printf("%s in %s at line %d\n", hipGetErrorString(err),
               file, line );
        exit(EXIT_FAILURE);
    }
}
#define cudaCheck( err ) (handleError(err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in, int *out) 
{

 // index of a thread across all threads + RADIUS
    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
    
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
        result += in[gindex + offset];

    // Store the result
    out[gindex - RADIUS] = result;
}

int main()
{
  	unsigned int i;

	hipEvent_t start, stop; //time start and stop
	float time;

	hipEventCreate(&start);
	hipEventCreate(&stop);

  //CPU array copies
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];

  // GPU array copies
  int *d_in, *d_out;

  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i )
    h_in[i] = 1; 

  // Allocate device memory
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  //copy fro CPU to GPU memory
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), 
  	hipMemcpyHostToDevice) );

  hipEventRecord( start, 0 );
  // Call stencil kernel
  stencil_1d<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);

	hipEventRecord( stop, 0 );
	hipEventSynchronize(stop);
	hipEventElapsedTime( &time, start, stop );
	hipEventDestroy( start );
	hipEventDestroy( stop );
	printf(" GPU Execution Time = %f\n",time);

  // Copy results from device memory to host
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int), 
  	hipMemcpyDeviceToHost) );

  
  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
